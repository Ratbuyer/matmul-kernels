#include <iostream>
#include <cassert>

#include "tools.cuh"
#include "1_naive.cuh"
#include "2_tiling.cuh"
#include "3_coalesce.cuh"
#include "4_vector.cuh"
#include "5_double_buffer.cuh"

constexpr int expected_argc = 7;

int main(int argc, char **argv) {
	
	if (argc != expected_argc) {
		std::cerr << "Usage: " << argv[0] << " <M> <K> <N> <Iterations> <check>" << std::endl;
		return 1;
	}
	
	const int M = std::stoi(argv[1]);
	const int N = std::stoi(argv[2]);
	const int K = std::stoi(argv[3]);
	const int iterations = std::stoi(argv[4]);
	const int kernel = std::stoi(argv[5]);
	const int check = std::stoi(argv[6]);
	
	if (M <= 0 || N <= 0 || K <= 0 || iterations <= 0 || kernel <= 0 || kernel > 6 || check < 0 || check > 1) {
		std::cerr << "Usage: " << argv[0] << " <M> <K> <N> <Iterations> <check>" << std::endl;
		return 1;
	}
	
	assert(M % 16 == 0);
	assert(K % 16 == 0);
	assert(N % 16 == 0);
	
	half *h_A = new half[M * K];
	half *h_B = new half[K * N];
	half *h_C = new half[M * N];
	
	half *d_A = nullptr;
	half *d_B = nullptr;
	half *d_C = nullptr;
	
	hipMalloc(&d_A, M * K * sizeof(half));
	hipMalloc(&d_B, K * N * sizeof(half));
	hipMalloc(&d_C, M * N * sizeof(half));
	
	fill_random(h_A, M, K);
	fill_random(h_B, K, N);
	
	hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice);

	// time kernel
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsed_time;
	
	hipEventRecord(start);
	
	for (int i = 0; i < iterations; i++) {
		switch (kernel) {
			case 1:
				launch_kernel_1(d_A, d_B, d_C, M, N, K);
				break;
			case 2:
				launch_kernel_2(d_A, d_B, d_C, M, N, K);
				break;
			case 3:
				launch_kernel_3(d_A, d_B, d_C, M, N, K);
				break;
			case 4:
				launch_kernel_4(d_A, d_B, d_C, M, N, K);
				break;
			case 5:
				launch_kernel_5(d_A, d_B, d_C, M, N, K);
				break;
			default:
				std::cerr << "Invalid kernel" << std::endl;
				return 1;
		}
	}
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	
	// check errors
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "CUDA error: " << hipGetErrorString(error)
            		<< " (Error code: " << static_cast<int>(error) << ")" << std::endl;
		return 1;
	}
	
	// printf("Latency: %f ms\n", elapsed_time / iterations);
	
	long long throughput = static_cast<long long>(M) * N * K * 2 * iterations;
    printf("Kernel: %d, M/N/K: %d, %d, %d, Throughput: %f GFLOPs/s\n", kernel, M, N, K, throughput * 1.0 / (elapsed_time * 1e6));
	
	// copy result back
	hipMemcpy(h_C, d_C, M * N * sizeof(half), hipMemcpyDeviceToHost);
	
	// print_matrix(h_C, M, N);
	
	half *cpu_C = new half[M * N];
	
	if (check) {
		CPU_gemm(h_A, h_B, cpu_C, M, N, K);
		compare_matrices(h_C, cpu_C, M, N);
	}
	
	// use this for debugging
	// print_differnce(h_C, cpu_C, M, N, 0.0);
	
	// free
	delete[] h_A;
	delete[] h_B;
	delete[] h_C;
	delete[] cpu_C;
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}